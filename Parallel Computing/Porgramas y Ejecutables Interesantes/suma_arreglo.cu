#include <stdio.h>
#include <hip/hip_runtime.h>

// Función del kernel que suma los elementos del arreglo
__global__ void suma_arreglo(float *a, size_t tam) {
    __shared__ float suma_par;

    size_t gtid = blockIdx.x * blockDim.x + threadIdx.x;	//global id
    size_t tid  = threadIdx.x;		// thread id, dentro del bloque
    size_t lid  = tid%warpSize;		// lane id, dentro del warp

    // Fase 1, inicialización
    if (tid==0)
    	suma_par = 0.0f;
    __syncthreads();

    // Fase 2, cómputo dentro del bloque
    float warp_reduce = a[gtid];

    // Fase 2.1, suma en warp
    #define FULL_MASK 0xffffffff
    warp_reduce += __shfl_down_sync(FULL_MASK, warp_reduce, 16);
    warp_reduce += __shfl_down_sync(FULL_MASK, warp_reduce, 8);
    warp_reduce += __shfl_down_sync(FULL_MASK, warp_reduce, 4);
    warp_reduce += __shfl_down_sync(FULL_MASK, warp_reduce, 2);
    warp_reduce += __shfl_down_sync(FULL_MASK, warp_reduce, 1);

    // Fase 2.2, acumulacion a shared
    if (lid==0) {
	atomicAdd(&suma_par, warp_reduce);
    }
    __syncthreads();

    // Fase 3, acumulación del resultado local del bloque en la global
    if (tid==0)
    	atomicAdd(&a[0], suma_par);
}



int main() {
    size_t tam = 1L<<31; // Tamaño del arreglo

    float *d_a;
    hipMalloc((void **)&d_a, tam * sizeof(float));

    // Configuramos los bloques y hilos para la ejecución del kernel
    suma_arreglo<<<tam/128L, 128>>>(d_a, tam);

    // Esperamos a que el kernel termine
    hipDeviceSynchronize();
    float suma_total;
    hipMemcpy(&suma_total, d_a, sizeof(float), hipMemcpyDeviceToHost);


    // Imprimimos la suma total
    printf("La suma total es: %f\n", suma_total);

    // Liberamos la memoria
    hipFree(d_a);
    return 0;
}
