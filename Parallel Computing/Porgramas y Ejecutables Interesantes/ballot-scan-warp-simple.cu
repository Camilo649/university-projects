#include "hip/hip_runtime.h"


//************************************
// TODO, no anda
//************************************

#include <stdio.h> // printf
#include <stdlib.h> // calloc
#include <assert.h> // assert
#include <hip/hip_runtime.h> // CUDA library
#include "hip/hip_runtime_api.h" // CUDA error checking

// integer ceiling division
#define DIV_CEIL(a,b) (((a)+(b)-1)/(b))
// maximum
#define MIN(a,b) (((a)<(b))?(a):(b))

#define BLOCK_SIZE 32
#define NUM_WARPS (BLOCK_SIZE/CUDA_WARP_SIZE)

__global__ void set(const uint N, int * const values) {
	int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	if (gtid<N)
		values[gtid] = (int) warpSize*(1+cosf((float)gtid));
}

__device__ __forceinline__ uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" :
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}

__global__ void ballotScanWarp(const int * const dataIn_global, int * const dataOut_global, uint * const countOut_global) {
	uint tid  = threadIdx.x;
	uint gtid = blockIdx.x*blockDim.x + threadIdx.x;

	int val = dataIn_global[gtid];

	uint flag = (42<=val);

	#define FULL_MASK 0xffffffff
	uint bits = __ballot_sync(FULL_MASK, flag);

	uint mask = bfi(0, FULL_MASK, 0, tid);
	uint exc = __popc(mask & bits);
	uint total = __popc(bits);

	dataOut_global[gtid] = exc;

	*countOut_global = total;
}


int main(int argc, char **argv)
{
	int *d_values = NULL, *h_values = NULL; // pointers in Host and Device
	int *d_inclusive = NULL, *h_inclusive = NULL;
	uint *d_count = NULL, *h_count = NULL;
	assert(2 == argc);
	const unsigned int N = (1<<atoi(argv[1])); // vector size
	assert(0 == N%CUDA_WARP_SIZE);

	checkCudaErrors(hipMalloc(&d_values, N * sizeof(int)));
	checkCudaErrors(hipMalloc(&d_inclusive, N * sizeof(int)));
	checkCudaErrors(hipMalloc(&d_count, 1 * sizeof(uint)));
	checkCudaErrors(hipHostMalloc(&h_values, N * sizeof(int)));
	checkCudaErrors(hipHostMalloc(&h_inclusive, N * sizeof(int)));
	checkCudaErrors(hipHostMalloc(&h_count, 1 * sizeof(uint)));
	assert(d_values && h_values && d_inclusive && h_inclusive && d_count && h_count);

	dim3 block_size(BLOCK_SIZE);
	dim3 grid_size(DIV_CEIL(N,BLOCK_SIZE));
	set<<<grid_size, block_size>>>(N, d_values);
	getLastCudaError("set kernel failed");
	ballotScanWarp<<<grid_size, block_size>>>(d_values, d_inclusive, d_count);
	getLastCudaError("ballotScanWarp kernel failed");
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(h_values, d_values, N*sizeof(int), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(h_count, d_count, 1*sizeof(int), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(h_inclusive, d_inclusive, (*h_count)*sizeof(int), hipMemcpyDefault));
	unsigned int i=0;
	// First warp
	printf("%d\n", *h_count);
	for (i=0; i<CUDA_WARP_SIZE; ++i)
		printf("%2d ", h_values[i]);
	printf("\n");
	for (i=0; i<*h_count; ++i)
		printf("%2d ", h_inclusive[i]);
	printf("\n");

	checkCudaErrors(hipHostFree(h_values));
	checkCudaErrors(hipHostFree(h_inclusive));
	checkCudaErrors(hipHostFree(h_count));
	checkCudaErrors(hipFree(d_values));
	checkCudaErrors(hipFree(d_inclusive));
	checkCudaErrors(hipFree(d_count));

	return 0;
}
