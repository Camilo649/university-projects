#include "hip/hip_runtime.h"
#include <stdio.h> // printf
#include <stdlib.h> // calloc
#include <assert.h> // assert
#include <hip/hip_runtime.h> // CUDA library
#include "hip/hip_runtime_api.h" // CUDA error checking

// integer ceiling division
#define DIV_CEIL(a,b) (((a)+(b)-1)/(b))
// minimum
#define MIN(a,b) (((a)<(b))?(a):(b))

#define BLOCK_SIZE 128
#define NUM_WARPS (BLOCK_SIZE/CUDA_WARP_SIZE)

__global__ void set(const uint N, int * const values) {
	int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	if (gtid<N)
		values[gtid] = (int) warpSize*(1+cosf((float)gtid));
}

__global__ void warp_scan3(const int * const values, int * const inclusive, int * const exclusive) {
	int tid = threadIdx.x;
	int lane = tid % warpSize;
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	// read from global
	int x = values[gtid];

	int sum = x;
	int n=0;
	#define FULL_MASK 0xffffffff
	n = __shfl_up_sync(FULL_MASK, sum, 1);
	if (lane >= 1) sum += n;
	__syncwarp();
	n = __shfl_up_sync(FULL_MASK, sum, 2);
	if (lane >= 2) sum += n;
	__syncwarp();
	n = __shfl_up_sync(FULL_MASK, sum, 4);
	if (lane >= 4) sum += n;
	__syncwarp();
	n = __shfl_up_sync(FULL_MASK, sum, 8);
	if (lane >= 8) sum += n;
	__syncwarp();
	n = __shfl_up_sync(FULL_MASK, sum, 16);
	if (lane >= 16) sum += n;
	__syncwarp();

	/*
	//Versión más cortita y de igual desempeño
	#pragma unroll
	for (int i=1; i<warpSize; i*=2) {
		int n = __shfl_up(FULL_MASK, sum, i);
		if (lane >= i) sum += n;
		__syncwarp();
	}
	*/
	

	inclusive[gtid] = sum;
	exclusive[gtid] = sum - x;
}



int main(int argc, char **argv)
{
	int *d_values = NULL, *h_values = NULL; // pointers in Host and Device
	int *d_inclusive = NULL, *h_inclusive = NULL;
	int *d_exclusive = NULL, *h_exclusive = NULL;
	assert(2 == argc);
	const unsigned int N = 1<<(atoi(argv[1])); // vector size
	assert(0 == N%32); // múltiplo de warp

	checkCudaErrors(hipMalloc(&d_values, N * sizeof(int)));
	checkCudaErrors(hipMalloc(&d_inclusive, N * sizeof(int)));
	checkCudaErrors(hipMalloc(&d_exclusive, N * sizeof(int)));
	checkCudaErrors(hipHostMalloc(&h_values, N * sizeof(int)));
	checkCudaErrors(hipHostMalloc(&h_inclusive, N * sizeof(int)));
	checkCudaErrors(hipHostMalloc(&h_exclusive, N * sizeof(int)));
	assert(d_values && h_values && d_inclusive && h_inclusive && d_exclusive && h_exclusive);

	dim3 block_size(BLOCK_SIZE);
	dim3 grid_size(DIV_CEIL(N,BLOCK_SIZE));
	set<<<grid_size, block_size>>>(N, d_values);
	getLastCudaError("set kernel failed");
	warp_scan3<<<grid_size, block_size>>>((const int*)d_values, d_inclusive, d_exclusive);
	getLastCudaError("warp_scan3 kernel failed");
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(h_values, d_values, N*sizeof(int), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(h_inclusive, d_inclusive, N*sizeof(int), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(h_exclusive, d_exclusive, N*sizeof(int), hipMemcpyDefault));
	unsigned int i=0;
	unsigned int lane=0;
	for (i=0; i<N; i+=CUDA_WARP_SIZE) {
		int inclusive = 0, exclusive = 0;
		for (lane=0; lane<CUDA_WARP_SIZE; ++lane) {
			inclusive += h_values[i+lane];
			if (inclusive!=h_inclusive[i+lane] || exclusive!=h_exclusive[i+lane]) {
				goto EXIT; // "Go To Statement Considered Harmful"
			}
			exclusive += h_values[i+lane];
		}
	}
EXIT:
	if (N != i) { // if error, print complete lane.
		printf("Error in warp %d, lane %d\n", i/CUDA_WARP_SIZE, lane);
		for (unsigned int lane=0; lane<CUDA_WARP_SIZE; ++lane) {
			printf("%d ", h_values[i+lane]);
		} printf("\n");
		for (unsigned int lane=0; lane<CUDA_WARP_SIZE; ++lane) {
			printf("%d ", h_inclusive[i+lane]);
		} printf("\n");
		for (unsigned int lane=0; lane<CUDA_WARP_SIZE; ++lane) {
			printf("%d ", h_exclusive[i+lane]);
		} printf("\n");
	}

	checkCudaErrors(hipHostFree(h_values));
	checkCudaErrors(hipHostFree(h_inclusive));
	checkCudaErrors(hipHostFree(h_exclusive));
	checkCudaErrors(hipFree(d_values));
	checkCudaErrors(hipFree(d_inclusive));
	checkCudaErrors(hipFree(d_exclusive));

	return 0;
}
